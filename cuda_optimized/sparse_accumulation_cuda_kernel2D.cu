#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <iostream>
using namespace torch::indexing;

#define BLOCK_SIZE 8

template <typename scalar_t>
__global__ void sparse_accumulation_cuda_forward_kernel(
    scalar_t* __restrict__ output,
    const scalar_t* __restrict__ X1,
    const scalar_t* __restrict__ X2,
    const int64_t* __restrict__ idx_output,
    const int64_t* __restrict__ idx_1,
    const int64_t* __restrict__ idx_2,
    const scalar_t* __restrict__ multipliers,
    const int32_t output_size,
    const int32_t X1_third_size,
    const int32_t X2_third_size,
    const int32_t nx,
    const int32_t ny,
    const int32_t nz
) {
    extern __shared__ char buffer[];
    // offset (in bytes) of the first available slot in the shared memory buffer
    size_t offset = 0;

    scalar_t* buffer_X1 = reinterpret_cast<scalar_t*>(buffer + offset);
    offset += BLOCK_SIZE * BLOCK_SIZE * X1_third_size * sizeof(scalar_t);

    scalar_t* buffer_X2 = reinterpret_cast<scalar_t*>(buffer + offset);
    offset += BLOCK_SIZE * BLOCK_SIZE * X2_third_size * sizeof(scalar_t);

    scalar_t* buffer_multipliers = reinterpret_cast<scalar_t*>(buffer + offset);
    offset += nz * sizeof(scalar_t);

    int32_t* buffer_idx_output = reinterpret_cast<int32_t*>(buffer + offset);
    offset += nz * sizeof(int32_t);

    int32_t* buffer_idx_X1 = reinterpret_cast<int32_t*>(buffer + offset);
    offset += nz * sizeof(int32_t);

    int32_t* buffer_idx_X2 = reinterpret_cast<int32_t*>(buffer + offset);
    offset += nz * sizeof(int32_t);

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    int single_multipliers_block_size = (nz / (BLOCK_SIZE * BLOCK_SIZE)) + 1;
    int total_thread_idx = threadIdx.x * BLOCK_SIZE + threadIdx.y;
    int multipliers_pos_from = total_thread_idx * single_multipliers_block_size;
    int multipliers_pos_to = (total_thread_idx + 1) * single_multipliers_block_size;
    if (multipliers_pos_to > nz) {
        multipliers_pos_to = nz;
    }

    int delta_now_X1 = j * X1_third_size + i * ny * X1_third_size;
    int delta_now_output = j * output_size + i * ny * output_size;
    int delta_now_X2 = j * X2_third_size + i * ny * X2_third_size;

   
    int delta_buffer_X1 = (BLOCK_SIZE * threadIdx.x + threadIdx.y) * X1_third_size;
    int delta_buffer_X2 = (BLOCK_SIZE * threadIdx.x + threadIdx.y) * X2_third_size;

    for (int active_index = multipliers_pos_from; active_index < multipliers_pos_to; ++active_index) {
        buffer_multipliers[active_index] = multipliers[active_index];
        buffer_idx_output[active_index] = idx_output[active_index];
        buffer_idx_X1[active_index] = idx_1[active_index];
        buffer_idx_X2[active_index] = idx_2[active_index];
    }
    
    scalar_t* buffer_X1_final = buffer_X1 + delta_buffer_X1;
    scalar_t* buffer_X2_final = buffer_X2 + delta_buffer_X2;

    auto output_final = output + delta_now_output;
    auto X1_final = X1 + delta_now_X1;
    auto X2_final = X2 + delta_now_X2;
    __syncthreads();

    if (i < nx && j < ny) {

        for (int X1_index = 0; X1_index < X1_third_size; ++X1_index) {
            buffer_X1_final[X1_index] = X1_final[X1_index];
        }

        for (int X2_index = 0; X2_index < X2_third_size; ++X2_index) {
            buffer_X2_final[X2_index] = X2_final[X2_index];
        }

        int z_output, z_X1, z_X2;
        scalar_t now = 0;
        int z_old = 0;
        for (int z = 0 ; z < nz ; ++z){
            z_output = buffer_idx_output[z];
            if (z_old != z_output) {
                output_final[z_old] = now;
                now = 0;
                z_old = z_output;
            }
            z_X1 = buffer_idx_X1[z];
            z_X2 = buffer_idx_X2[z];
            now += buffer_X1_final[z_X1]
                                           * buffer_X2_final[z_X2]
                                           * buffer_multipliers[z];
        };
        output_final[z_old] = now;
    };
}

template <typename scalar_t>
__global__ void sparse_accumulation_cuda_backward_kernel(
    scalar_t* __restrict__ d_X1,
    scalar_t* __restrict__ d_X2,
    const scalar_t* __restrict__ d_output,
    const scalar_t* __restrict__ X1,
    const scalar_t* __restrict__ X2,
    const int64_t* __restrict__ idx_output,
    const int64_t* __restrict__ idx_1,
    const int64_t* __restrict__ idx_2,
    const scalar_t* __restrict__ multipliers,
    const int output_size,
    const int X1_third_size,
    const int X2_third_size,
    const int nx,
    const int ny,
    const int nz
    ) {
    extern __shared__ char buffer[];
    // offset (in bytes) of the first available slot in the shared memory buffer
    size_t offset = 0;
    scalar_t* buffer_output = reinterpret_cast<scalar_t*>(buffer + offset);
    offset += BLOCK_SIZE * BLOCK_SIZE * output_size * sizeof(scalar_t);

    scalar_t* buffer_X1 = reinterpret_cast<scalar_t*>(buffer + offset);
    offset += BLOCK_SIZE * BLOCK_SIZE * X1_third_size * sizeof(scalar_t);

    scalar_t* buffer_X2 = reinterpret_cast<scalar_t*>(buffer + offset);
    offset += BLOCK_SIZE * BLOCK_SIZE * X2_third_size * sizeof(scalar_t);

    scalar_t* buffer_d_X1 = reinterpret_cast<scalar_t*>(buffer + offset);
    offset += BLOCK_SIZE * BLOCK_SIZE * X1_third_size * sizeof(scalar_t);

    scalar_t* buffer_d_X2 = reinterpret_cast<scalar_t*>(buffer + offset);
    offset += BLOCK_SIZE * BLOCK_SIZE * X2_third_size * sizeof(scalar_t);

    scalar_t* buffer_multipliers = reinterpret_cast<scalar_t*>(buffer + offset);
    offset += nz * sizeof(scalar_t);

    int32_t* buffer_idx_output = reinterpret_cast<int32_t*>(buffer + offset);
    offset += nz * sizeof(int32_t);

    int32_t* buffer_idx_X1 = reinterpret_cast<int32_t*>(buffer + offset);
    offset += nz * sizeof(int32_t);

    int32_t* buffer_idx_X2 = reinterpret_cast<int32_t*>(buffer + offset);
    offset += nz * sizeof(int32_t);

    int i = threadIdx.x + blockDim.x * blockIdx.x;
    int j = threadIdx.y + blockDim.y * blockIdx.y;

    int single_multipliers_block_size = (nz / (BLOCK_SIZE * BLOCK_SIZE)) + 1;
    int total_thread_idx = threadIdx.x * BLOCK_SIZE + threadIdx.y;
    int multipliers_pos_from = total_thread_idx * single_multipliers_block_size;
    int multipliers_pos_to = (total_thread_idx + 1) * single_multipliers_block_size;
    if (multipliers_pos_to > nz) {
        multipliers_pos_to = nz;
    }

    int delta_now_X1 = j * X1_third_size + i * ny * X1_third_size;
    int delta_now_output = j * output_size + i * ny * output_size;
    int delta_now_X2 = j * X2_third_size + i * ny * X2_third_size;
    int delta_now_d_X1 = j * X1_third_size + i * ny * X1_third_size;
    int delta_now_d_X2 = j * X2_third_size + i * ny * X2_third_size;

    int delta_buffer_output = (BLOCK_SIZE * threadIdx.x + threadIdx.y) * output_size;
    int delta_buffer_X1 = (BLOCK_SIZE * threadIdx.x + threadIdx.y) * X1_third_size;
    int delta_buffer_X2 = (BLOCK_SIZE * threadIdx.x + threadIdx.y) * X2_third_size;
    int delta_buffer_d_X1 = (BLOCK_SIZE * threadIdx.x + threadIdx.y) * X1_third_size;
    int delta_buffer_d_X2 = (BLOCK_SIZE * threadIdx.x + threadIdx.y) * X2_third_size;


    for (int active_index = multipliers_pos_from; active_index < multipliers_pos_to; ++active_index) {
        buffer_multipliers[active_index] = multipliers[active_index];
        buffer_idx_output[active_index] = idx_output[active_index];
        buffer_idx_X1[active_index] = idx_1[active_index];
        buffer_idx_X2[active_index] = idx_2[active_index];
    }
    scalar_t* buffer_output_final = buffer_output + delta_buffer_output;
    scalar_t* buffer_X1_final = buffer_X1 + delta_buffer_X1;
    scalar_t* buffer_X2_final = buffer_X2 + delta_buffer_X2;
    scalar_t* buffer_d_X1_final = buffer_d_X1 + delta_buffer_d_X1;
    scalar_t* buffer_d_X2_final = buffer_d_X2 + delta_buffer_d_X2;

    auto d_output_final = d_output + delta_now_output;
    auto X1_final = X1 + delta_now_X1;
    auto X2_final = X2 + delta_now_X2;
    auto d_X1_final = d_X1 + delta_now_X1;
    auto d_X2_final = d_X2 + delta_now_X2;
    __syncthreads();



    // int i = threadIdx.x + blockDim.x * blockIdx.x ;
    // int j = threadIdx.y + blockDim.y * blockIdx.y ;

    // if (i<nx && j<ny) {
    //   for (auto z = 0 ; z < nz ; ++z){
    //     int z_output = idx_output[z];
    //     int z_X1 = idx_1[z] ;
    //     int z_X2 = idx_2[z] ;

    //     int pos_X1 = z_X1 + j*X1_third_size + i*ny*X1_third_size ;
    //     int pos_output = z_output+ j*output_size+  i*output_size*ny ;
    //     int pos_X2 = z_X2 + j*X2_third_size + i*ny*X2_third_size ;
    //     auto grad_multi = d_output[pos_output] * multipliers[z];
    //     d_X1[pos_X1] += grad_multi*X2[pos_X2];
    //     d_X2[pos_X2] += grad_multi*X1[pos_X1];
    //   };
    // };
 if (i < nx && j < ny) {
        //printf("in kernel i %d  j %d\n",i,j) ;
        for (int z_output = 0; z_output < output_size; ++z_output) {
            buffer_output_final[z_output] = d_output_final[z_output];
        }

        for (int X1_index = 0; X1_index < X1_third_size; ++X1_index) {
            buffer_X1_final[X1_index] = X1_final[X1_index];
            buffer_d_X1_final[X1_index] = 0;
        }

        for (int X2_index = 0; X2_index < X2_third_size; ++X2_index) {
            buffer_X2_final[X2_index] = X2_final[X2_index];
            buffer_d_X2_final[X2_index] = 0;
        }

        int z_output, z_X1, z_X2;
        //scalar_t now = 0;
        int z_old = 0;
        scalar_t grad_multi;
        for (int z = 0 ; z < nz ; ++z){
            z_output = buffer_idx_output[z];
            // if (z_old != z_output) {
            //     output_final[z_old] = now;
            //     now = 0;
            //     z_old = z_output;
            // }
            z_X1 = buffer_idx_X1[z];
            z_X2 = buffer_idx_X2[z];
            grad_multi = buffer_output_final[z_output] * buffer_multipliers[z];
            buffer_d_X1_final[z_X1] += grad_multi * buffer_X2_final[z_X2];
                                           
            buffer_d_X2_final[z_X2] += grad_multi * buffer_X1_final[z_X1];
        };
        //output_final[z_old] = now;
        for (int z = 0; z < X1_third_size; ++z) {
            d_X1_final[z] = buffer_d_X1_final[z];
        }

        for (int z = 0; z < X2_third_size; ++z) {
            d_X2_final[z] = buffer_d_X2_final[z];
        }
        /*for (int z = 0 ; z < nz ; ++z){
            z_X1 = buffer_idx_X1[z];
            z_X2 = buffer_idx_X2[z];
            d_X2_final[z_X2] = buffer_d_X2_final[z_X2];
            d_X1_final[z_X1] = buffer_d_X1_final[z_X1];
        }*/
        

    };


  }


std::vector<torch::Tensor> sparse_accumulation_cuda_forward(
    torch::Tensor X1,
    torch::Tensor X2,
    torch::Tensor idx_output,
    int output_size,
    torch::Tensor idx_1,
    torch::Tensor idx_2,
    torch::Tensor multipliers
) {
  //auto output = torch::zeros_like(X1);
  auto output = torch::zeros({X1.sizes()[0], X1.sizes()[1], output_size},
            torch::TensorOptions()
            .dtype(X1.dtype())
            .device(X1.device()));

  auto X1_third_size = X1.sizes()[2];
  auto X2_third_size = X2.sizes()[2];
  const auto batch_sizex = output.sizes()[0];
  const auto batch_sizey = output.sizes()[1];
  const auto batch_sizez = idx_output.sizes()[0];

  auto nx = batch_sizex ;
  auto ny = batch_sizey ;
  auto nz = batch_sizez ;
  // auto threads = 124;
  //const dim3 blocks((n+threads-1)/threads, batch_size);
  //auto blocks = (n+threads-1)/threads;

  //AT_DISPATCH_FLOATING_TYPES(output.type(), "sparse_accumulation_forward_cuda", ([&] {
  //  sparse_accumulation_cuda_forward_kernel<scalar_t><<<blocks, threads>>>(
  //      output.data<scalar_t>(),
  //      X1.data<scalar_t>(),
  //      n1,
  //      n2,
  //      );
  //}));

  auto find_num_blocks = [](int x, int bdim) {return (x+bdim-1)/bdim;};
  dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE);
  int nbx = find_num_blocks(nx, block_dim.x);
  int nby = find_num_blocks(ny, block_dim.y);
  int nbz = find_num_blocks(nz, block_dim.z);
  dim3 grid_dim(nbx, nby);


  AT_DISPATCH_FLOATING_TYPES(output.type(), "sparse_accumulation_forward_cuda", ([&] {
      size_t X1_buf_size = BLOCK_SIZE * BLOCK_SIZE * X1_third_size * sizeof(scalar_t);
      size_t X2_buf_size = BLOCK_SIZE * BLOCK_SIZE * X2_third_size * sizeof(scalar_t);
      size_t multipliers_size = multipliers.sizes()[0] * sizeof(scalar_t);
      size_t index_size = idx_output.sizes()[0] * sizeof(int32_t);

      size_t total_buf_size = X1_buf_size + X2_buf_size + multipliers_size + index_size * 3;

      sparse_accumulation_cuda_forward_kernel<<<grid_dim, block_dim, total_buf_size>>>(
          output.data_ptr<scalar_t>(),
          X1.data_ptr<scalar_t>(),
          X2.data_ptr<scalar_t>(),
          idx_output.data_ptr<int64_t>(),
          idx_1.data_ptr<int64_t>(),
          idx_2.data_ptr<int64_t>(),
          multipliers.data_ptr<scalar_t>(),
          output_size,
          X1_third_size,
          X2_third_size,
          nx,
          ny,
          nz
      );
  }));

  return {output};
}

std::vector<torch::Tensor> sparse_accumulation_cuda_backward(
    torch::Tensor d_output,
    torch::Tensor X1,
    torch::Tensor X2,
    torch::Tensor idx_output,
    torch::Tensor idx_1,
    torch::Tensor idx_2,
    torch::Tensor multipliers
) {
    auto d_X1 = torch::zeros_like(X1);
    auto d_X2 = torch::zeros_like(X2);

    auto X1_third_size = X1.sizes()[2];
    auto X2_third_size = X2.sizes()[2];
    const auto nx = d_output.sizes()[0]    ;
    const auto ny = d_output.sizes()[1]    ;
    const auto output_size = d_output.sizes()[2] ;
    const auto nz = idx_output.sizes()[0];

    auto find_num_blocks = [](int x, int bdim) {return (x+bdim-1)/bdim;};
    dim3 block_dim(BLOCK_SIZE, BLOCK_SIZE);
    int nbx = find_num_blocks(nx, block_dim.x);
    int nby = find_num_blocks(ny, block_dim.y);
    dim3 grid_dim(nbx, nby);

    AT_DISPATCH_FLOATING_TYPES(X1.type(), "sparse_accumulation_backward_cuda", ([&] {
        size_t output_buf_size = BLOCK_SIZE * BLOCK_SIZE * output_size * sizeof(scalar_t);
        size_t X1_buf_size = BLOCK_SIZE * BLOCK_SIZE * X1_third_size * sizeof(scalar_t);
        size_t X2_buf_size = BLOCK_SIZE * BLOCK_SIZE * X2_third_size * sizeof(scalar_t);
        size_t multipliers_size = multipliers.sizes()[0] * sizeof(scalar_t);
        size_t index_size = idx_output.sizes()[0] * sizeof(int32_t);

        size_t total_buf_size = output_buf_size + 2*X1_buf_size + 2*X2_buf_size + multipliers_size + index_size * 3;
        sparse_accumulation_cuda_backward_kernel<<<grid_dim, block_dim,total_buf_size>>>(
            d_X1.data_ptr<scalar_t>(),
            d_X2.data_ptr<scalar_t>(),
            d_output.data_ptr<scalar_t>(),
            X1.data_ptr<scalar_t>(),
            X2.data_ptr<scalar_t>(),
            idx_output.data_ptr<int64_t>(),
            idx_1.data_ptr<int64_t>(),
            idx_2.data_ptr<int64_t>(),
            multipliers.data_ptr<scalar_t>(),
            output_size,
            X1_third_size,
            X2_third_size,
            nx,
            ny,
            nz
        );
    }));

    return {d_X1, d_X2};
}

#define CHECK_CUDA(x) TORCH_CHECK(x.device().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

std::vector<torch::Tensor> sparse_accumulation_gpu_forward(
    torch::Tensor X1,
    torch::Tensor X2,
    torch::Tensor idx_output,
    int64_t output_size,
    torch::Tensor idx_1,
    torch::Tensor idx_2,
    torch::Tensor multipliers
) {
    CHECK_INPUT(X1);
    CHECK_INPUT(X2);
    CHECK_INPUT(idx_output);
    //CHECK_INPUT(output_size);
    CHECK_INPUT(idx_1);
    CHECK_INPUT(idx_2);
    CHECK_INPUT(multipliers);

    return sparse_accumulation_cuda_forward(X1,X2,idx_output,output_size,idx_1,idx_2,multipliers);
}

std::vector<torch::Tensor> sparse_accumulation_gpu_backward(
    torch::Tensor d_output,
    torch::Tensor X1,
    torch::Tensor X2,
    torch::Tensor idx_output,
    torch::Tensor idx_1,
    torch::Tensor idx_2,
    torch::Tensor multipliers
) {
    CHECK_INPUT(d_output);
    CHECK_INPUT(X1);
    CHECK_INPUT(X2);
    CHECK_INPUT(idx_output);
    CHECK_INPUT(idx_1);
    CHECK_INPUT(idx_2 );
    CHECK_INPUT(multipliers);

    return sparse_accumulation_cuda_backward(d_output,X1,X2,idx_output,idx_1,idx_2,multipliers);
}

TORCH_LIBRARY(sparse_accumulation_cuda, m) {
    m.def("forward", sparse_accumulation_gpu_forward);
    m.def("backward", sparse_accumulation_gpu_backward);
}
